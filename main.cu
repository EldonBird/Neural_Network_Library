#include "hip/hip_runtime.h"
#include "Network.h"


__global__ void Vectoraddition(int* a, int* b, int* c, int n){

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if(i < n){
        c[i] = a[i] + b[i];
    }
}

__global__ void calculate_outputs(double* last_activation, double* last_weight_inputs; double** weights, double* biases, double* last_inputs, double* inputs, double* outputs, int size_in, int size_out){

    for(int i = 0; i < size_in; i++){
        last_inputs[i] = inputs[i];
    }

    for(int x = 0; x < size_out; x++){
        double weighted_input = biases[x];
        for(int y = 0; y < size_in; y++){
            weighted_input += inputs[y] * weights[x][y];
        }
        last_weighted_inputs[x] = weighted_input;
        last_activation[x] = 1.0 / (1.0 + (-weighted_input * -weighted_input));
    }
}


int main(){

    int N = 10;
    size_t size = N * sizeof(float);


    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    Vectoraddition<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);



    return 1;
}